/*
Baseline 4 : Thuật toán Radix Sort tuần tự theo hướng dẫn đồ án
Cài đặt mảng lưu bin như file hướng dẫn. (hàng là các block, cột là các bin)
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Sequential Radix Sort
void sortByHost(const uint32_t * in, int n,
                uint32_t * out)
{

    int nBits = 4; // Assume: nBits in {1, 2, 4, 8, 16}
    int nBins = 1 << nBits; // 2^nBits

    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    // Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
    // (Each digit consists of nBits bit)
    // In each loop, sort elements according to the current digit from src to dst 
    // (using STABLE counting sort)
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        // TODO: Compute histogram
        memset(hist, 0, nBins * sizeof(int));
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            hist[bin]++;
        }

        // TODO: Scan histogram (exclusively)
        histScan[0] = 0;
        for (int bin = 1; bin < nBins; bin++)
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];

        // TODO: Scatter elements to correct locations
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            dst[histScan[bin]] = src[i];
            histScan[bin]++;
        }
        
        // Swap src and dst
        uint32_t * temp = src;
        src = dst;
        dst = temp;
    }

    // Copy result to out
   memcpy(out, src, n * sizeof(uint32_t)); 
}

// Parallel Radix Sort
void sortByDevice(const uint32_t * in, int n, uint32_t * out, int bklSize)
{
    // TODO
    int nBits = 1; // Assume: nBits in {1, 2, 4, 8, 16}
    int nBins = 1 << nBits; // 2^nBits

    dim3 blockSize(bklSize); // block size
    dim3 gridSize((n - 1) / blockSize.x + 1); // grid size

    int * hist = (int *)malloc(nBins * gridSize.x * sizeof(int));
    int *histScan = (int * )malloc(nBins * gridSize.x * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        memset(hist, 0, nBins * gridSize.x * sizeof(int));
        for (int i = 0; i < gridSize.x; i++)
        {
            for (int j = 0; j < blockSize.x; j++)
            if (i * blockSize.x + j < n)
            {
                int bin = (src[i * blockSize.x + j] >> bit) & (nBins - 1);
                hist[i * nBins + bin]++;
            }
        }

        int previous = 0;
        for (int j = 0; j < nBins; j++){
            for (int i = 0; i < gridSize.x; i++)
            {
                histScan[i * nBins + j] = previous;
                previous = previous + hist[i * nBins + j];
            }
        }

        for (int i = 0; i < gridSize.x; i++)
        {
            for (int j = 0; j < blockSize.x; j++)
            {
                int id = i * blockSize.x + j;
                if (id < n)
                {
                    int bin = i * nBins + ((src[id] >> bit) & (nBins - 1));
                    dst[histScan[bin]] = src[id];
                    histScan[bin]++;
                }
            }
        }
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }

    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    free(hist);
    free(histScan);
}

// Radix Sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        bool useDevice=false, int blockSize=1)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
        printf("\nRadix Sort by host\n");
        sortByHost(in, n, out);
    }
    else // use device
    {
        printf("\nRadix Sort by device\n");
        sortByDevice(in, n, out, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 24) + 1; // For test by eye
    //int n = (1 << 24) + 1;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
    {
        in[i] = rand() % 255; // For test by eye
        //in[i] = rand();
    }
    // printArray(in, n); // For test by eye

    // DETERMINE BLOCK SIZE
    int blockSize = 512; // Default 
    if (argc == 2)
        blockSize = atoi(argv[1]);

    // SORT BY HOST
    sort(in, n, correctOut);
    // printArray(correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, out, true, blockSize);
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
