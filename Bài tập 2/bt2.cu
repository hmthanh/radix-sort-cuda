
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__global__ void reduceBlksKernel1(int * in, int n, int * out)
{
	// TODO
}

__global__ void reduceBlksKernel2(int * in, int n, int * out)
{
	// TODO
}

__global__ void reduceBlksKernel3(int * in, int n, int * out)
{
	// TODO
}

int reduce(int const * in, int n,
			bool useDevice=false, dim3 blockSize=dim3(1), int kernelType=1)
{

	int result = 0; // Init
	if (useDevice == false)
	{
		result = in[0];
		for (int i = 1; i < n; i++)
			result += in[i];
	}
	else // Use device
	{
		// Allocate device memories
		int * d_in, * d_out;
		dim3 gridSize(1); // TODO: Compute gridSize from n and blockSize
		CHECK(hipMalloc(&d_in, n * sizeof(int)));
		CHECK(hipMalloc(&d_out, gridSize.x * sizeof(int)));

		// Copy data to device memory
		CHECK(hipMemcpy(d_in, in, n*sizeof(int), hipMemcpyHostToDevice));

		// Call kernel
		GpuTimer timer;
		timer.Start();
		if (kernelType == 1)
			reduceBlksKernel1<<<gridSize, blockSize>>>(d_in, n, d_out);
		else if (kernelType == 2)
			reduceBlksKernel2<<<gridSize, blockSize>>>(d_in, n, d_out);
		else
			reduceBlksKernel3<<<gridSize, blockSize>>>(d_in, n, d_out);
		timer.Stop();
		float kernelTime = timer.Elapsed();
		hipDeviceSynchronize();
		CHECK(hipGetLastError());

		// Copy result from device memory
		int * out = (int *)malloc(gridSize.x * sizeof(int));
		CHECK(hipMemcpy(out, d_out, gridSize.x*sizeof(int), hipMemcpyDeviceToHost));

		// Free device memories
		CHECK(hipFree(d_in));
		CHECK(hipFree(d_out));

		// Host do the rest of the work
		timer.Start();
		result = out[0];
		for (int i = 1; i < gridSize.x; i++)
		{
			result += out[i];
		}
		timer.Stop();
		float postKernelTime = timer.Elapsed();

		// Free memory
		free(out);

		// Print info
		printf("\nKernel %d\n", kernelType);
		printf("Grid size: %d, block size: %d\n", gridSize.x, blockSize.x);
		printf("Kernel time = %f ms, post-kernel time = %f ms\n", kernelTime, postKernelTime);
	}

	return result;
}

void checkCorrectness(int r1, int r2)
{
	if (r1 == r2)
		printf("CORRECT :)\n");
	else
		printf("INCORRECT :(\n");
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}
int main(int argc, char ** argv)
{
	printDeviceInfo();

	// Set up input size
    int n = (1 << 24) + 1;
    printf("Input size: %d\n", n);

    // Set up input data
    int * in = (int *) malloc(n * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        // Generate a random integer in [0, 255]
        in[i] = (int)(rand() & 0xFF);
    }

    // Reduce NOT using device
    int correctResult = reduce(in, n);

    // Reduce using device, kernel1
    dim3 blockSize(512); // Default
    if (argc == 2)
    	blockSize.x = atoi(argv[1]);
    int result1 = reduce(in, n, true, blockSize, 1);
    checkCorrectness(result1, correctResult);

    // Reduce using device, kernel2
    int result2 = reduce(in, n, true, blockSize, 2);
    checkCorrectness(result2, correctResult);

    // Reduce using device, kernel3
    int result3 = reduce(in, n, true, blockSize, 3);
    checkCorrectness(result3, correctResult);

    // Free memories
    free(in);
}