/*
Baseline 4.2 : Song song 2 bước tính hist và scan*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// Sequential Radix Sort
void sortByHost(const uint32_t * in, int n,
                uint32_t * out)
{

    int nBits = 4; // Assume: nBits in {1, 2, 4, 8, 16}
    int nBins = 1 << nBits; // 2^nBits

    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    // Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
    // (Each digit consists of nBits bit)
    // In each loop, sort elements according to the current digit from src to dst 
    // (using STABLE counting sort)
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        // TODO: Compute histogram
        memset(hist, 0, nBins * sizeof(int));
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            hist[bin]++;
        }

        // TODO: Scan histogram (exclusively)
        histScan[0] = 0;
        for (int bin = 1; bin < nBins; bin++)
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];

        // TODO: Scatter elements to correct locations
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            dst[histScan[bin]] = src[i];
            histScan[bin]++;
        }
        
        // Swap src and dst
        uint32_t * temp = src;
        src = dst;
        dst = temp;
    }

    // Copy result to out
   memcpy(out, src, n * sizeof(uint32_t)); 
}

// #########################################################
// Baseline
void sortBaseline(const uint32_t * in, int n, uint32_t * out, int bklSize)
{
    // TODO
    int nBits = 1; // Assume: nBits in {1, 2, 4, 8, 16}
    int nBins = 1 << nBits; // 2^nBits

    dim3 blockSize(bklSize); // block size
    dim3 gridSize((n - 1) / blockSize.x + 1); // grid size

    int * hist = (int *)malloc(nBins * gridSize.x * sizeof(int));
    int *histScan = (int * )malloc(nBins * gridSize.x * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        memset(hist, 0, nBins * gridSize.x * sizeof(int));
        // compute historgram
        for (int i = 0; i < gridSize.x; i++)
        {
            for (int j = 0; j < blockSize.x; j++)
            if (i * blockSize.x + j < n)
            {
                int bin = (src[i * blockSize.x + j] >> bit) & (nBins - 1);
                hist[i * nBins + bin]++;
            }
        }

        // compute scan
        int previous = 0;
        for (int j = 0; j < nBins; j++){
            for (int i = 0; i < gridSize.x; i++)
            {
                histScan[i * nBins + j] = previous;
                previous = previous + hist[i * nBins + j];
            }
        }

        // scatter
        for (int i = 0; i < gridSize.x; i++)
        {
            for (int j = 0; j < blockSize.x; j++)
            {
                int id = i * blockSize.x + j;
                if (id < n)
                {
                    int bin = i * nBins + ((src[id] >> bit) & (nBins - 1));
                    dst[histScan[bin]] = src[id];
                    histScan[bin]++;
                }
            }
        }
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }

    memcpy(out, src, n * sizeof(uint32_t));
    free(hist);
    free(histScan);
}

// #########################################################
// Radix sort by device
// #########################################################
// Histogram kernel
__global__ void computeHistogram(uint32_t * in, int n, int * hist, int nBins, int bit)
{
    // TODO
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        int bin = (in[i] >> bit) & (nBins - 1);
        atomicAdd(&hist[bin * gridDim.x + blockIdx.x], 1);
    }
}

// scan kernel
__global__ void scanExclusiveBlk(int * in, int n, int * out, int * blkSums)
{   
    // TODO
    extern __shared__ int s_data[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < n){
        s_data[threadIdx.x] = in[i - 1];
    }
    else{
        s_data[threadIdx.x] = 0;
    }
    __syncthreads();
    
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int val = 0;
        if (threadIdx.x >= stride){
            val = s_data[threadIdx.x - stride];
        }
        __syncthreads();

        s_data[threadIdx.x] += val;
        __syncthreads();
    }
    
    if (i < n){
        out[i] = s_data[threadIdx.x];
    }
    if (blkSums != NULL){
        blkSums[blockIdx.x] = s_data[blockDim.x - 1];
    }
}

__global__ void computeHistScan(int * in, int n, int* blkSums)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && blockIdx.x > 0)
        in[i] += blkSums[blockIdx.x - 1];
}

// Parallel Radix Sort
void sortByDevice(const uint32_t * in, int n, uint32_t * out, int bklSize)
{
    // TODO
    int nBits = 1; // Assume: nBits in {1, 2, 4, 8, 16}
    int nBins = 1 << nBits; // 2^nBits

    dim3 blockSize(bklSize);
    dim3 gridHistSize((n - 1) / blockSize.x + 1);
    dim3 gridScanSize((nBins * gridHistSize.x - 1) / blockSize.x + 1);
    
    int * scan = (int * )malloc(nBins * gridHistSize.x * sizeof(int));
    int * blkSums = (int *)malloc(gridScanSize.x * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * dst = out;

    uint32_t * d_src;
    int *d_hist, *d_scan, *d_blkSums;

    CHECK(hipMalloc(&d_src, n * sizeof(uint32_t)));
    CHECK(hipMalloc(&d_hist, nBins * gridHistSize.x * sizeof(int)));
    CHECK(hipMalloc(&d_scan, nBins * gridHistSize.x * sizeof(int)));
    CHECK(hipMalloc(&d_blkSums, gridScanSize.x * sizeof(int)));

    size_t smemHistBytes = nBins * sizeof(int); 
    size_t smemScanBytes = blockSize.x * sizeof(int);
    
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        // compute historgram
        CHECK(hipMemcpy(d_src, src, n * sizeof(uint32_t), hipMemcpyHostToDevice));
        CHECK(hipMemset(d_hist, 0, nBins * gridHistSize.x * sizeof(int)));
        computeHistogram<<<gridHistSize, blockSize, smemHistBytes>>>(d_src, n, d_hist, nBins, bit);
        hipDeviceSynchronize();

        // compute scan
        scanExclusiveBlk<<<gridScanSize, blockSize, smemScanBytes>>>(d_hist, nBins * gridHistSize.x, d_scan, d_blkSums);
        hipDeviceSynchronize();
        
        CHECK(hipMemcpy(blkSums, d_blkSums, gridScanSize.x * sizeof(int), hipMemcpyDeviceToHost));
        for (int i = 1; i < gridScanSize.x; i++){
            blkSums[i] += blkSums[i - 1];
        }
        CHECK(hipMemcpy(d_blkSums, blkSums, gridScanSize.x * sizeof(int), hipMemcpyHostToDevice));
        computeHistScan<<<gridScanSize, blockSize>>>(d_scan, nBins * gridHistSize.x, d_blkSums);
        hipDeviceSynchronize();
        CHECK(hipMemcpy(scan, d_scan, nBins * gridHistSize.x * sizeof(int), hipMemcpyDeviceToHost)); 
        
        // Scatter
        for (int i = 0; i < n ; i++)
        {
            int bin = i / blockSize.x + ((src[i] >> bit) & (nBins - 1)) * gridHistSize.x;
            dst[scan[bin]] = src[i];
            scan[bin]++;
        }
        uint32_t * temp = src;
        src = dst;
        dst = temp;
    }
    // Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));

    CHECK(hipFree(d_src));
    CHECK(hipFree(d_hist));
    CHECK(hipFree(d_scan));
    CHECK(hipFree(d_blkSums));
    
    free(blkSums);
    free(scan);
}

// Radix Sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        bool useDevice=false, int blockSize=1, int type=0)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
        printf("\nRadix Sort by host\n");
        sortByHost(in, n, out);
    }
    else if (type == 1){ // Baseline
        printf("\nBaseline Radix Sort (highlight)\n");
        sortBaseline(in, n, out, blockSize);
    }
    else // use device
    {
        printf("\nRadix Sort by device\n");
        sortByDevice(in, n, out, blockSize);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 24) + 1; // For test by eye
    //int n = (1 << 24) + 1;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out_baseline = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
    {
        in[i] = rand() % 255; // For test by eye
        //in[i] = rand();
    }
    // printArray(in, n); // For test by eye

    // DETERMINE BLOCK SIZE
    int blockSize = 512; // Default 
    if (argc == 2)
        blockSize = atoi(argv[1]);

    // SORT BY HOST
    sort(in, n, correctOut);
    // printArray(correctOut, n);

    // SORT BY BASELINE
    sort(in, n, out_baseline, true, blockSize, 1);
    checkCorrectness(out_baseline, correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, out, true, blockSize, 2);
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
